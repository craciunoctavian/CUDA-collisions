#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "CheckCollision.cuh"
#include "Box.h"

#define cudaCheckError() { \
	hipError_t e=hipGetLastError(); \
	if(e!=hipSuccess) { \
		printf("Cuda failure, %s",hipGetErrorString(e)); \
		exit(0); \
	 }\
}

__global__ 
void kernel_check(int noOfCubes, float* d_x, float* d_y, float* d_z, float* d_yVel)
{
	int idxX = blockIdx.x * blockDim.x + threadIdx.x;
	int idxY = blockIdx.y * blockDim.y + threadIdx.y;
	if (idxX > noOfCubes || idxY > noOfCubes) {
		return;
	}
	float c_x = d_x[idxX];
	float c_y = d_y[idxX];
	float c_z = d_z[idxX];

	float b_x = d_x[idxY];
	float b_y = d_y[idxY];
	float b_z = d_z[idxY];

	if (idxX == idxY) return;
	// AABB collision
	if ((c_x - 0.5 <= b_x + 0.5 && c_x + 0.5 >= b_x - 0.5) &&
		(c_y - 0.5 <= b_y + 0.5 && c_y + 0.5 >= b_y - 0.5) &&
		(c_z - 0.5 <= b_z + 0.5 && c_z + 0.5 >= b_z - 0.5)) {
		if (c_y > b_y) {
			d_yVel[idxX] = 2.0f;
		}
	}
}

bool CUDA::checkCollision() {

	float *h_x; 
	float *h_y; 
	float *h_z; 
	float *h_yVel;

	h_x = (float*)malloc(noOfCubes * sizeof(float));
	h_y = (float*)malloc(noOfCubes * sizeof(float));
	h_z = (float*)malloc(noOfCubes * sizeof(float));
	h_yVel = (float*)malloc(noOfCubes * sizeof(float));

	for (int i = 0; i < noOfCubes; i++) {
		h_x[i] = boxes[i].x;
		h_y[i] = boxes[i].y;
		h_z[i] = boxes[i].z;
		h_yVel[i] = boxes[i].yVel;
	}

	float *d_x;
	float *d_y;
	float *d_z;
	float *d_yVel;

	hipMalloc((void**)&d_x, noOfCubes * sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&d_y, noOfCubes * sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&d_z, noOfCubes * sizeof(float));
	cudaCheckError();
	hipMalloc((void**)&d_yVel, noOfCubes * sizeof(float));
	cudaCheckError();

	hipMemcpy(d_x, h_x, noOfCubes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, noOfCubes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_z, h_z, noOfCubes * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_yVel, h_yVel, noOfCubes * sizeof(float), hipMemcpyHostToDevice);

	dim3 blocksPerGrid((noOfCubes + 15) / 16, (noOfCubes + 15) / 16, 1);
	dim3 threadsPerBlock(16, 16, 1);
	kernel_check<<<blocksPerGrid, threadsPerBlock >>>(noOfCubes, d_x, d_y, d_z, d_yVel);

	// Wait for GPU to finish before accessing on host
	hipDeviceSynchronize();

	hipMemcpy(h_yVel, d_yVel, noOfCubes * sizeof(float), hipMemcpyDeviceToHost);

	for (int i = 0; i < noOfCubes; i++) {
		boxes[i].yVel = h_yVel[i];
	}

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_z);
	hipFree(d_yVel);
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_yVel);

	return true;
}
