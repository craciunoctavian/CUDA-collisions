#include "hip/hip_runtime.h"
#include "Box.h"
#include <numeric>
#include <vector>
#include <algorithm>
#include <random>
#include <set>
#include "CheckCollision.cuh"

#define M_PI 3.141592f

float velF = 0.1f;
float gravF = 0.005f;
float velyF = 0.1f;
std::map<int,box> boxes;
int noOfCubes = 500;
int id = 0;
float g = 9.8f;
float D = 5.5f;

// random float between 0 and 1
float randomFloat() {
	return float(rand()) / float((RAND_MAX));
}

void Animations::generateCube(float x, float y, float z) {
	float xVel = sin(randomFloat() * 2 * M_PI);
	float zVel = cos(randomFloat() * 2 * M_PI);
	boxes[id] = box{id, randomFloat() * 9 + 1, x, y, z, xVel, zVel, 0, 0, 
		glm::vec3(randomFloat(), randomFloat(), randomFloat()) };
	id++;
}

void Animations::initCubes() {
	srand((unsigned int)time(NULL));
	float x, y, z;
	for (int i = 0; i < noOfCubes; i++) {
		while (true) {
			bool collision = false;
			// generate random coordinates for cube
			x = float(rand()) / float((RAND_MAX)) * 22 - 11;
			y = float(rand()) / float((RAND_MAX)) * 22 - 11;
			z = float(rand()) / float((RAND_MAX)) * 22 - 11;
			// check to be in the extra box all the time
			if (abs(sqrt(x * x + z * z)) > 10) continue;

			// check collision at generation of cube
			for (int i = 0; i < noOfCubes; i++) {
				// we have collision between two boxes
				if (abs(x - boxes[i].x) < 1 && abs(y - boxes[i].y) < 1 && abs(z - boxes[i].z) < 1)
				{
					collision = true;
					break;
				}
			}
			if (!collision) break;
		}
		generateCube(x, y, z);
	}
}


void checkCollisionBigBox(box b) {

	float yPos = b.y + 1 - (12);
	float yNeg = b.y - (-12);
	float zPos = b.z + 1 - (12);
	float zNeg = b.z - (-12);
	float xPos = b.x + 1 - (12);
	float xNeg = b.x - (-12);

	// collision with extraBox bottom
	if ((yNeg < 0.5) && b.moving) {
		boxes[b.id].moving = false;
	}

	// collision with extraBox top
	if (yPos > 0.5) {
		boxes[b.id].timeFalling = 0;
		boxes[b.id].yVel = 0;
	}

	// collision with extraBox sides
	if (zPos > 0.5 || zNeg < 0.5 || xPos > 0.5 || xNeg < 0.5) {
		boxes[b.id].zVel = -boxes[b.id].zVel;
		boxes[b.id].xVel = -boxes[b.id].xVel;
	}
}


void Animations::moveCubes(float deltaTimeSeconds) {

	for (int i = 0; i < noOfCubes; i++) {
		if (!boxes[i].moving) continue;
		boxes[i].timeFalling += deltaTimeSeconds;
	}
	
	CUDA::checkCollision();

	for (int i = 0; i < noOfCubes; i++) {
		checkCollisionBigBox(boxes[i]);
		if (!boxes[i].moving) continue;

		float A = ((g - (float)(D / boxes[i].m)) / 2.0f)
			* (float)(2.0f * boxes[i].timeFalling + 1.0f) * gravF;
		float bump = boxes[i].yVel * velyF;

		boxes[i].y -= A - bump;

		if (boxes[i].yVel > 0) {
			boxes[i].yVel -= 0.1;
			if (A > bump) {
				boxes[i].timeFalling = 0;
				boxes[i].yVel = 0;
			}
		}

		boxes[i].x += boxes[i].xVel * velF;
		boxes[i].z += boxes[i].zVel * velF;
	}
}
